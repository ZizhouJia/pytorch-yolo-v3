#include "hip/hip_runtime.h"
#include "nms_cuda.h"

__device__ int get_index(){
  int blockId=blockIdx.y*gridDim.x+blockIdx.x;
  int threadId=blockId*blockDim.x+threadIdx.x;
  return threadId;
}

__device__ int get_block_prefix(){
    int blockId=blockIdx.y*gridDim.x+blockIdx.x;
    int block_prefix=blockId*blockDim.x;
    return block_prefix;
}

__device__ int get_thread(){
  return threadIdx.x;
}

__global__ void nms_cuda_imp(float* bbox, int64_t* bbox_size,
   float* mask,int64_t* mask_size,float thresh){
     int index=get_index();
     int block_prefix=get_block_prefix();
     for(int i=0;i<bbox_size[1];++i){
       __syncthreads();
       if(index>=bbox_size[0]*bbox_size[1]){
         continue;
       }
       if(i>=get_thread()){
         continue;
       }
       if(mask[block_prefix+i]==0){
         continue;
       }
       if(mask[index]==0){
         continue;
       }
       if(bbox[7*(block_prefix+i)+6]!=bbox[7*(index)+6]){
         continue;
       }
       float x11=bbox[7*(block_prefix+i)+0];
       float y11=bbox[7*(block_prefix+i)+1];
       float x12=bbox[7*(block_prefix+i)+2];
       float y12=bbox[7*(block_prefix+i)+3];
       float x21=bbox[7*(index)+0];
       float y21=bbox[7*(index)+1];
       float x22=bbox[7*(index)+2];
       float y22=bbox[7*(index)+3];
       float areas_u=(x12-x11)*(y12-y11)+(x22-x21)*(y22-y21);
       float max_x1=((x11>=x21)?x11:x21);
       float max_y1=((y11>=y21)?y11:y21);
       float min_x2=((x12<=x22)?x12:x22);
       float min_y2=((y12<=y22)?y12:y22);
       float w=min_x2-max_x1;
       w=(w>=0?w:0);
       float h=min_y2-max_y1;
       h=(h>=0?h:0);
       float areas_n=w*h;
       if(areas_u-areas_n==0){
         continue;
       }
       float iou=areas_n/(areas_u-areas_n);
       if(iou>thresh){
         mask[index]=0;
       }
     }
   }

void cuda_cpy(int64_t* from,int64_t** to,int size){
  hipMalloc((void**)to,size*sizeof(int64_t));
  hipMemcpy(*to,from,size*sizeof(int64_t),hipMemcpyHostToDevice);
}

void nms_cuda(float* bbox, int64_t* bbox_size,
   float* mask,int64_t* mask_size,float thresh,hipStream_t stream){
  int d1=1;
  int d2=1;
  if(bbox_size[0]>512){
    d2=(bbox_size[0]+511)/512;
    d1=512;
  }else{
    d1=bbox_size[0];
    d2=1;
  }
  dim3 batch(d1,d2,1);
  dim3 thread(bbox_size[1],1,1);
  int64_t* bbox_size_cuda;
  int64_t* mask_size_cuda;
  cuda_cpy(bbox_size,&bbox_size_cuda,3);
  cuda_cpy(mask_size,&mask_size_cuda,2);
  nms_cuda_imp<<<batch,thread,0,stream>>>(bbox,bbox_size_cuda,mask,mask_size_cuda,thresh);

  hipFree(bbox_size_cuda);
  hipFree(mask_size_cuda);
}
